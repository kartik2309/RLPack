#include "hip/hip_runtime.h"

#pragma clang diagnostic push
#pragma ide diagnostic ignored "misc-no-recursion"
#include "C_ReplayBuffer.cuh"


C_ReplayBuffer::C_ReplayBuffer(int64_t bufferSize,
                               const std::string &device,
                               const int32_t &prioritizationStrategyCode,
                               const int32_t &batchSize) {
    /*!
     * The class constructor for C_ReplayBuffer. This constructor initialised the C_ReplayBuffer class and allocates the
     * required memory as per input arguments. This initialises the rlpack._C.replay_buffer.ReplayBuffer.C_ReplayBuffer and is
     * equivalent to rlpack._C.replay_buffer.ReplayBuffer.__init__.
     *
     * @param bufferSize : The buffer size to be used and allocated for the memory.
     * @param device : The device transfer relevant tensors to.
     * @param prioritizationStrategyCode : The prioritization strategy code. Refer
     *  rlpack.dqn.dqn.Dqn.get_prioritization_code.
     * @param batchSize : The batch size to be used for sampling.
     *
     */
    bufferSize_ = bufferSize;
    device_ = Maps::deviceMap[device];
    prioritizationStrategyCode_ = prioritizationStrategyCode;
    batchSize_ = batchSize;
    cMemoryData = std::make_shared<C_ReplayBufferData>();
    auto statesCurrentRawPointer = &statesCurrent_;
    auto statesNextRawPointer = &statesNext_;
    auto rewardsRawPointer = &rewards_;
    auto actionsRawPointer = &actions_;
    auto donesRawPointer = &dones_;
    auto prioritiesRawPointer = &priorities_;
    auto probabilitiesRawPointer = &probabilities_;
    auto weightsRawPointer = &weights_;
    cMemoryData->set_transition_information_references(statesCurrentRawPointer,
                                                       statesNextRawPointer,
                                                       rewardsRawPointer,
                                                       actionsRawPointer,
                                                       donesRawPointer,
                                                       prioritiesRawPointer,
                                                       probabilitiesRawPointer,
                                                       weightsRawPointer);
    auto terminalStateIndicesRawPointer = &terminalStateIndices_;
    cMemoryData->set_terminal_state_indices_reference(terminalStateIndicesRawPointer);
    auto prioritiesFloatRawPointer = &prioritiesFloat_;
    cMemoryData->set_priorities_reference(prioritiesFloatRawPointer);
    loadedIndices_.reserve(bufferSize_);
    sumTreeSharedPtr_ = nullptr;
    switch (prioritizationStrategyCode_) {
        case 1:
            sumTreeSharedPtr_ = std::make_shared<SumTree>(bufferSize_);
            break;
        case 2:
            segmentQuantileIndices_ = std::vector<int64_t>(batchSize_);
            break;
        default:
            break;
    }
    offloadFloat_ = new Offload<float_t>(bufferSize_);
    offloadInt64_ = new Offload<int64_t>(bufferSize_);
    loadedIndicesSlice_ = std::vector<int64_t>(batchSize_);
    seedValues_ = std::vector<float_t>(bufferSize_);
    sampledStateCurrent_ = std::vector<torch::Tensor>(batchSize_);
    sampledStateNext_ = std::vector<torch::Tensor>(batchSize_);
    sampledRewards_ = std::vector<torch::Tensor>(batchSize_);
    sampledActions_ = std::vector<torch::Tensor>(batchSize_);
    sampledDones_ = std::vector<torch::Tensor>(batchSize_);
    sampledPriorities_ = std::vector<torch::Tensor>(batchSize_);
    sampledIndices_ = std::vector<torch::Tensor>(batchSize_);
}

C_ReplayBuffer::C_ReplayBuffer() {
    /*!
     * The default non-parameterised constructor. This constructor allocates memory as per default initialised variables.
     * This initialises the rlpack._C.replay_buffer.ReplayBuffer.C_ReplayBuffer and is equivalent to rlpack._C.replay_buffer.ReplayBuffer.__init__.
     */
    cMemoryData = std::make_shared<C_ReplayBufferData>();
    auto statesCurrentRawPointer = &statesCurrent_;
    auto statesNextRawPointer = &statesNext_;
    auto rewardsRawPointer = &rewards_;
    auto actionsRawPointer = &actions_;
    auto donesRawPointer = &dones_;
    auto prioritiesRawPointer = &priorities_;
    auto probabilitiesRawPointer = &probabilities_;
    auto weightsRawPointer = &weights_;
    cMemoryData->set_transition_information_references(statesCurrentRawPointer,
                                                       statesNextRawPointer,
                                                       rewardsRawPointer,
                                                       actionsRawPointer,
                                                       donesRawPointer,
                                                       prioritiesRawPointer,
                                                       probabilitiesRawPointer,
                                                       weightsRawPointer);
    auto terminalStateIndicesRawPointer = &terminalStateIndices_;
    cMemoryData->set_terminal_state_indices_reference(terminalStateIndicesRawPointer);
    auto prioritiesFloatRawPointer = &prioritiesFloat_;
    cMemoryData->set_priorities_reference(prioritiesFloatRawPointer);
    loadedIndices_.reserve(bufferSize_);
    sumTreeSharedPtr_ = nullptr;
    switch (prioritizationStrategyCode_) {
        case 1:
            sumTreeSharedPtr_ = std::make_shared<SumTree>(bufferSize_);
            break;
        case 2:
            segmentQuantileIndices_ = std::vector<int64_t>(batchSize_);
            break;
        default:
            break;
    }
    offloadFloat_ = new Offload<float_t>(bufferSize_);
    offloadInt64_ = new Offload<int64_t>(bufferSize_);
    loadedIndicesSlice_ = std::vector<int64_t>(batchSize_);
    seedValues_ = std::vector<float_t>(bufferSize_);
}


C_ReplayBuffer::~C_ReplayBuffer() {
    /*!
     * The destructor for C_ReplayBuffer.
     */
    delete offloadFloat_;
    delete offloadInt64_;
}

void C_ReplayBuffer::insert(torch::Tensor &stateCurrent,
                            torch::Tensor &stateNext,
                            torch::Tensor &reward,
                            torch::Tensor &action,
                            torch::Tensor &done,
                            torch::Tensor &priority,
                            torch::Tensor &probability,
                            torch::Tensor &weight,
                            bool isTerminalState) {
    /*!
     * Insertion method for C_ReplayBuffer. This is the C++ backend of rlpack._C.replay_buffer.ReplayBuffer.insert method.
     *
     * @param stateCurrent : Current state from transition
     * @param stateNext : Next state from transition.
     * @param reward : Reward obtained during transition.
     * @param action : Action taken during transition.
     * @param done : Flag indicating if next state is terminal packaged in PyTorch Tensor.
     * @param priority : Priority value associated with the transition.
     * @param probability : Probability value associated with the transition.
     * @param weight : Weight value associated with the transition.
     * @param isTerminalState : Flag indicating if next state is terminal.
     */
    if (size() > bufferSize_) {
        delete_item(0);
    }
    statesCurrent_.push_back(stateCurrent);
    statesNext_.push_back(stateNext);
    rewards_.push_back(reward);
    actions_.push_back(action);
    dones_.push_back(done);
    priorities_.push_back(priority);
    probabilities_.push_back(probability);
    weights_.push_back(weight);
    prioritiesFloat_.push_back(priority.item<float_t>());
    if (size() < bufferSize_) {
        loadedIndices_.push_back(stepCounter_);
        stepCounter_ += 1;
    }
    if (isTerminalState) {
        terminalStateIndices_.push_back((int64_t) size() - 1);
    }
}

std::map<std::string, torch::Tensor> C_ReplayBuffer::get_item(int64_t index) {
    /*!
     * Getter method for C_ReplayBuffer. This is the C++ backend of rlpack._C.replay_buffer.ReplayBuffer.__getitem__ method so can be
     * accessed by simple indexing operation (with operator []; item = memory[index]) from Python side.
     *
     * @param index : The index from which we want to obtain the transition
     * @return A map of transition quantities. The map will contain the following keys:
     *  - states_current
     *  - states_next
     *  - rewards
     *  - actions
     *  - dones
     *  - priorities
     *  - probabilities
     *  - weights
     */
    if (index >= size()) {
        throw std::out_of_range("Index is larger than current size of memory!");
    }
    std::map<std::string, torch::Tensor> returnItems = {
            {"states_current", statesCurrent_[index]},
            {"states_next", statesNext_[index]},
            {"rewards", rewards_[index]},
            {"actions", actions_[index]},
            {"dones", dones_[index]},
            {"priorities", priorities_[index]},
            {"probabilities", probabilities_[index]},
            {"weights", weights_[index]},
    };
    return returnItems;
}

void C_ReplayBuffer::set_item(int64_t index,
                              torch::Tensor &stateCurrent,
                              torch::Tensor &stateNext,
                              torch::Tensor &reward,
                              torch::Tensor &action,
                              torch::Tensor &done,
                              torch::Tensor &priority,
                              torch::Tensor &probability,
                              torch::Tensor &weight,
                              bool isTerminalState) {
    /*!
     * Setter method for C_ReplayBuffer. This is the C++ backend of rlpack._C.replay_buffer.ReplayBuffer.__setitem__ method so can be
     * accessed by simple indexing operation (with operator []; memory[index] = index) from Python side.
     * This method modified the items at the given index.
     *
     * @param index : The index to which we want to set the transition.
     * @param stateCurrent : Current state from transition
     * @param stateNext : Next state from transition.
     * @param reward : Reward obtained during transition.
     * @param action : Action taken during transition.
     * @param done : Flag indicating if next state is terminal packaged in PyTorch Tensor.
     * @param priority : Priority value associated with the transition.
     * @param probability : Probability value associated with the transition.
     * @param weight : Weight value associated with the transition.
     * @param isTerminalState : Flag indicating if next state is terminal.
     */
    if (index >= size()) {
        throw std::out_of_range("Given index is larger than current size! Use insert method to expand the memory.");
    }
    statesCurrent_[index] = stateCurrent;
    statesNext_[index] = stateNext;
    rewards_[index] = reward;
    actions_[index] = action;
    dones_[index] = done;
    priorities_[index] = priority;
    probabilities_[index] = probability;
    weights_[index] = weight;
    if (isTerminalState) {
        auto findIter = std::find(
                terminalStateIndices_.begin(), terminalStateIndices_.end(), index);
        if (findIter != terminalStateIndices_.end()) {
            terminalStateIndices_.push_back(index);
        }
    }
    prioritiesFloat_[index] = priority.item<float_t>();
    loadedIndices_[index] = index;
    switch (prioritizationStrategyCode_) {
        case 1:
            sumTreeSharedPtr_->update(index, priority.item<float_t>());
        default:
            break;
    }
}

void C_ReplayBuffer::delete_item(int64_t index) {
    /*!
     * Deletion method for C_ReplayBuffer. This is the C++ backend of rlpack._C.replay_buffer.ReplayBuffer.__delitem__ so can be
     * accessed by simple indexing operation (with operator []; del memory[index]) from Python side.
     *
     *
     * This the deletion is fast if index is either the first or last element, else will take O(n) to allocate memory
     * for items after index.
     *
     * @param index : The index of the transition we want to remove.
     */
    if (index >= size()) {
        throw std::out_of_range("Index is larger than current size of memory!");
    }
    if (index != 0) {
        statesCurrent_.erase(statesCurrent_.begin() + index);
        statesNext_.erase(statesNext_.begin() + index);
        rewards_.erase(rewards_.begin() + index);
        actions_.erase(actions_.begin() + index);
        if (dones_[index].flatten().item<int32_t>() == 1) {
            auto indexIter = std::find(
                    terminalStateIndices_.begin(), terminalStateIndices_.end(), index);
            if (indexIter != terminalStateIndices_.end()) {
                terminalStateIndices_.erase(indexIter);
            } else {
                std::cerr << "WARNING: Deletion of terminal state occurred but "
                             "terminal state was not found in `terminalStateIndices_`"
                          << std::endl;
            }
        }
        dones_.erase(dones_.begin() + index);
        priorities_.erase(priorities_.begin() + index);
        probabilities_.erase(probabilities_.begin() + index);
        weights_.erase(weights_.begin() + index);
        prioritiesFloat_.erase(prioritiesFloat_.begin() + index);
    } else {
        statesCurrent_.pop_front();
        statesNext_.pop_front();
        rewards_.pop_front();
        actions_.pop_front();
        dones_.pop_front();
        priorities_.pop_front();
        probabilities_.pop_front();
        weights_.pop_front();
        if (dones_[0].flatten().item<int32_t>() == 1) {
            terminalStateIndices_.pop_front();
        }
        prioritiesFloat_.pop_front();
    }
}

std::map<std::string, torch::Tensor> C_ReplayBuffer::sample(float_t forceTerminalStateProbability,
                                                            int64_t parallelismSizeThreshold,
                                                            float_t alpha,
                                                            float_t beta,
                                                            int64_t numSegments) {
    /*!
     * The sampling method for C_ReplayBuffer. This is the C++ backend of rlpack._C.replay_buffer.ReplayBuffer.sample. Sampling is done
     * as per the prioritization strategy specified during initialisation of C_ReplayBuffer.
     *
     * @param forceTerminalStateProbability : The probability to force a terminal state in final sample.
     * @param parallelismSizeThreshold : The threshold size of buffer (from C_ReplayBuffer::size method) beyond with
     * OpenMP parallelized routines are used for sampling.
     * @param alpha : The alpha value for prioritization. This is used to compute probabilities, where higher alpha
     * indicates more aggressive prioritization.
     * @param beta : The beta value for prioritization. This is used to compute important sampling weights, where higher
     * beta indicates more aggressive bias correction.
     * @param numSegments : The number of segments to be used for rank-based prioritization (in accordance with Zipf's law)
     * @return A map of sampled transitions separated by quantities. The map has the following keys with
     * each key containing a tensor of shape `(batchSize, ...)`:
     *  - states_current
     *  - states_next
     *  - rewards
     *  - actions
     *  - dones
     *  - priorities
     *  - probabilities
     *  - weights
     */
    std::random_device rd;
    std::mt19937 generator(rd());
    std::uniform_real_distribution<float_t> distributionP(0, 1);
    std::uniform_int_distribution<int64_t> distributionOfTerminalIndex(0,
                                                                       (int64_t) terminalStateIndices_.size() - 1);

    int64_t index = 0;
    bool forceTerminalState = false;
    switch (prioritizationStrategyCode_) {
        case 0: {
            offloadInt64_->reset();
            offloadInt64_->shuffle(loadedIndices_, parallelismSizeThreshold);
            memcpy(&loadedIndicesSlice_[0], &offloadInt64_->result[0], sizeof(int64_t) * batchSize_);
            break;
        }
        case 1: {
            // Proportional prioritization sampling.
            {
                bool enableParallelism = parallelismSizeThreshold < prioritiesFloat_.size();
//        Parallel region to execute resetting and creation of tree simultaneously with getting priority seeds.
//        Two sections are spawned for each associated function.
//        Sections 0: Executes resetting and creation of tree.
//        Sections 1: Executes computation of cumulative sum and generates priority seeds.
#pragma omp parallel sections if (enableParallelism) default(none) \
        firstprivate(parallelismSizeThreshold, prioritiesFloat_)   \
                shared(sumTreeSharedPtr_, seedValues_)
                {
#pragma omp section
                    {
                        sumTreeSharedPtr_->reset(parallelismSizeThreshold);
                        std::optional<std::vector<SumTreeNode *>> nullOptVector = std::nullopt;
                        sumTreeSharedPtr_->create_tree(prioritiesFloat_, nullOptVector);
                    }
#pragma omp section
                    {
                        offloadFloat_->reset();
                        auto cumulativeSum = offloadFloat_->cumulative_sum(prioritiesFloat_,
                                                                           parallelismSizeThreshold);
                        if (seedValues_.size() < static_cast<size_t>(cumulativeSum)) {
                            seedValues_.resize(static_cast<size_t>(cumulativeSum));
                        }
                        offloadFloat_->reset();
                        offloadFloat_->generate_priority_seeds(cumulativeSum, parallelismSizeThreshold);
                        memcpy(&seedValues_[0], &offloadFloat_->result[0], sizeof(float_t) * batchSize_);
                    }
                }
            }
            for (int32_t batchIndex = 0; batchIndex < batchSize_; batchIndex++) {
                auto seedValue = seedValues_[batchIndex];
                auto randomIndex = sumTreeSharedPtr_->sample(seedValue,
                                                             (int64_t) size());
                loadedIndicesSlice_[batchIndex] = randomIndex;
            }
            break;
        }
        case 2: {
            // Rank-Based prioritization sampling.
            int64_t previousQuantileIndex = 0, generatedRandomIndex = 0;
            index = 0;
            std::uniform_int_distribution<int64_t> distributionOfSegment(-1, 1);
            offloadFloat_->reset();
            offloadFloat_->arg_quantile_segment_indices(numSegments,
                                                        prioritiesFloat_,
                                                        parallelismSizeThreshold);
            memcpy(&segmentQuantileIndices_[0], &offloadFloat_->result[0], sizeof(float_t) * batchSize_);
            for (auto &segmentQuantileIndex: segmentQuantileIndices_) {
                if ((segmentQuantileIndex - previousQuantileIndex - 1) <= 1) {
                    generatedRandomIndex = previousQuantileIndex;
                } else {
                    distributionOfSegment.reset();
                    distributionOfSegment.param(std::uniform_int_distribution<int64_t>::param_type(previousQuantileIndex,
                                                                                                   segmentQuantileIndex - 1));
                    generatedRandomIndex = distributionOfSegment(generator);
                }
                loadedIndicesSlice_[index] = loadedIndices_[generatedRandomIndex];
                previousQuantileIndex = segmentQuantileIndex;
                index++;
            }
            index = 0;
            break;
        }
        default:
            break;
    }
    float_t p = distributionP(generator);
    if (size() < batchSize_) {
        throw std::out_of_range("Requested batch size is larger than current Memory size!");
    }
    if (p < forceTerminalStateProbability && terminalStateIndices_.size() > 1) {
        forceTerminalState = true;
    }
    if (forceTerminalState) {
        int64_t randomIndexToInsertTerminalState = distributionOfTerminalIndex(generator) % batchSize_;
        int64_t randomTerminalStateInfoIndex = distributionOfTerminalIndex(generator);
        int64_t randomTerminalStateIndex = terminalStateIndices_[randomTerminalStateInfoIndex];
        loadedIndicesSlice_[randomIndexToInsertTerminalState] = randomTerminalStateIndex;
    }
    for (auto &loadedIndex: loadedIndicesSlice_) {
        sampledStateCurrent_[index] = statesCurrent_[loadedIndex];
        sampledStateNext_[index] = statesNext_[loadedIndex];
        sampledRewards_[index] = rewards_[loadedIndex];
        sampledActions_[index] = actions_[loadedIndex];
        sampledDones_[index] = dones_[loadedIndex];
        sampledPriorities_[index] = priorities_[loadedIndex];
        sampledIndices_[index] = torch::full({}, loadedIndex);
        sampledIndices_[index] = torch::full({}, loadedIndex);
        index++;
    }
    auto floatTensorOptions = torch::TensorOptions().device(device_).dtype(torch::kFloat32);
    auto int64TensorOptions = torch::TensorOptions().device(device_).dtype(torch::kInt64);
    auto statesCurrentStacked = torch::stack(sampledStateCurrent_, 0).to(floatTensorOptions);
    auto statesNextStacked = torch::stack(sampledStateNext_, 0).to(floatTensorOptions);
    auto rewardsStacked = torch::stack(sampledRewards_, 0).to(floatTensorOptions);
    auto actionsStacked = torch::stack(sampledActions_, 0).to(int64TensorOptions);
    auto donesStacked = torch::stack(sampledDones_, 0).to(floatTensorOptions);
    auto prioritiesStacked = torch::stack(sampledPriorities_, 0).to(floatTensorOptions);
    auto sampledIndicesStacked = torch::stack(sampledIndices_, 0).to(int64TensorOptions);
    std::map<std::string, torch::Tensor> samples = {
            {"states_current", statesCurrentStacked},
            {"states_next", statesNextStacked},
            {"rewards", rewardsStacked},
            {"actions", actionsStacked},
            {"dones", donesStacked},
            {"priorities", prioritiesStacked},
            {"random_indices", sampledIndicesStacked}};
    if (prioritizationStrategyCode_ != 0) {
        auto probabilities = compute_probabilities(prioritiesStacked, alpha).to(floatTensorOptions);
        auto weights = compute_important_sampling_weights(probabilities,
                                                          (int64_t) size(),
                                                          beta)
                               .to(floatTensorOptions);
        samples["probabilities"] = probabilities;
        samples["weights"] = weights;
    } else {
        samples["probabilities"] = torch::zeros(prioritiesStacked.sizes(), floatTensorOptions);
        samples["weights"] = torch::zeros(prioritiesStacked.sizes(), floatTensorOptions);
    }
    return samples;
}

void C_ReplayBuffer::update_priorities(torch::Tensor &randomIndices,
                                       torch::Tensor &newPriorities) {
    /*!
     * The method to update priorities as per new values computed by agent as per the prioritization strategy. This
     * is the C++ backend of rlpack._C.replay_buffer.ReplayBuffer.update_priorities method.
     *
     * @param randomIndices : The random indices on which priorities are required to be updated. C_ReplayBuffer::sample
     * provides this information which can be used.
     * @param newPriorities : The new priorities computed by the agent as per the prioritization strategy.
     */
    if (prioritizationStrategyCode_ == 0) {
        throw std::runtime_error("`update_priorities` method called in C++ backend when C_ReplayBuffer is un-prioritized!");
    }
    newPriorities = newPriorities.flatten();
    randomIndices = randomIndices.flatten();
    auto size = randomIndices.size(0);
    for (int32_t index = 0; index < size; index++) {
        auto selectIndex = randomIndices[index].item<int64_t>();
        priorities_[selectIndex] = newPriorities[index];
        prioritiesFloat_[selectIndex] = newPriorities[index].item<float_t>();
    }
}

C_ReplayBufferData C_ReplayBuffer::view() const {
    /*!
     * The pointer to C_ReplayBufferData object. This will contain references of data in C_ReplayBuffer and provides
     * an easy data view. This is the C++ backend of rlpack._C.replay_buffer.ReplayBuffer.view method.
     */
    return *cMemoryData;
}

void C_ReplayBuffer::initialize(C_ReplayBufferData &viewC_MemoryData) {
    /*!
     * Initialize method for C_ReplayBuffer for initializing all the data from an object of C_ReplayBufferData. This is
     * the C++ backend of rlpack._C.replay_buffer.ReplayBuffer.initialize method
     *
     * @param viewC_MemoryData : An object of C_ReplayBufferData from which C_ReplayBuffer has to be initialized.
     */
    cMemoryData = std::make_shared<C_ReplayBufferData>(viewC_MemoryData);
    auto transitionInformation = cMemoryData->dereference_transition_information();
    auto terminalStateIndices = cMemoryData->dereference_terminal_state_indices();
    auto prioritiesFloat = cMemoryData->dereference_priorities();
    statesCurrent_ = transitionInformation["states_current"];
    statesNext_ = transitionInformation["states_next"];
    rewards_ = transitionInformation["rewards"];
    actions_ = transitionInformation["actions"];
    dones_ = transitionInformation["dones"];
    priorities_ = transitionInformation["priorities"];
    probabilities_ = transitionInformation["probabilities"];
    weights_ = transitionInformation["weights"];
    std::vector<int64_t> loadedIndices(bufferSize_);
    for (int64_t index = 0; index < size(); index++) {
        loadedIndices[index] = index;
    }
    loadedIndices_ = loadedIndices;
    stepCounter_ = (int64_t) size();
}

void C_ReplayBuffer::clear() {
    /*!
     * Clears the data in C_ReplayBuffer. This will **NOT** free the memory since it doesn't perform any memory de-allocation.
     * This is C++ backend of rlpack._C.replay_buffer.ReplayBuffer.clear method.
     */
    statesCurrent_.clear();
    statesNext_.clear();
    rewards_.clear();
    actions_.clear();
    dones_.clear();
    priorities_.clear();
    probabilities_.clear();
    weights_.clear();
    if (sumTreeSharedPtr_ != nullptr) {
        sumTreeSharedPtr_->reset();
    }
}

size_t C_ReplayBuffer::size() {
    /*!
     * This method obtains the current size of C_ReplayBuffer. This is the C++ backend of rlpack._C.replay_buffer.ReplayBuffer.__len__
     * method, so length can be obtained by in-built python function len(memory).
     *
     * @return The size(or length) of C_ReplayBuffer.
     */
    return dones_.size();
}

int64_t C_ReplayBuffer::num_terminal_states() {
    /*!
     * Method to obtain the number of terminal states currently in C_ReplayBuffer. This is the C++ backend of
     * rlpack._C.replay_buffer.ReplayBuffer.num_terminal_states method.
     *
     * @return Number of terminal states so far.
     */
    return (int64_t) terminalStateIndices_.size();
}

int64_t C_ReplayBuffer::tree_height() {
    /*!
     * Method to obtain the tree height of the sum tree if using a proportional prioritization strategy. This is
     * the C++ backend of rlpack._C.replay_buffer.ReplayBuffer.tree_height. If not using proportional prioritization strategy,
     * calling this method will throw an error.
     *
     * @return The tree height of the tree built.
     */
    // sumTreeSharedPtr_ is set to nullptr by default and only changes when using proportional prioritization strategy.
    if (sumTreeSharedPtr_ == nullptr) {
        throw std::runtime_error("Accessing `tree_height` method when not using proportional prioritization strategy");
    }
    return sumTreeSharedPtr_->get_tree_height();
}

torch::Tensor C_ReplayBuffer::compute_probabilities(torch::Tensor &priorities, float_t alpha) {
    /*!
     * Method to compute probabilities when not using uniform prioritization strategy.
     *
     * @param priorities : The sampled priorities for which probabilities are to be computed.
     * @param alpha : The alpha value for prioritization. Refer C_ReplayBuffer::sample for more information.
     * @return The tensor with probabilities corresponding to each priority.
     */
    auto prioritiesPowered = torch::pow(priorities, alpha);
    auto probabilities = prioritiesPowered / torch::sum(prioritiesPowered);
    return probabilities;
}

torch::Tensor C_ReplayBuffer::compute_important_sampling_weights(torch::Tensor &probabilities,
                                                                 int64_t currentSize,
                                                                 float_t beta) {
    /*!
     * Method to compute the important sampling weights for each probabilities.
     *
     * @param probabilities : The input probabilities for which IS weights are to be computed.
     * @param currentSize : The current size of the C_ReplayBuffer (see C_ReplayBuffer::size)
     * @param beta : The beta value for prioritization. Refer C_ReplayBuffer::sample for more information.
     * @return The tensor with important sampling weights corresponding to each probability.
     */
    auto weights = torch::pow(1 / (currentSize * probabilities), beta);
    auto maxWeightInBatch = weights.max().item<float_t>();
    weights = weights / maxWeightInBatch;
    return weights;
}

#pragma clang diagnostic pop
